#include "hip/hip_runtime.h"
/*
CSC501 - Operating System - Spring 2012 - North Carolina State University

HomeWork2 Prob4. See - http://courses.ncsu.edu/csc501/lec/001/hw/hw2/
Author: Salil Kanitkar (sskanitk@ncsu.edu)

For Compiling - 
$ make clean ; make a4
For Executing -
$ ./a4 <path-to-log-file> <path-to-process-list-file>
*/

#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include <hip/hip_runtime.h>
#include<sys/types.h>
#include <math.h>

#ifndef _WIN32
#include<sys/time.h>
#endif

/* Uncomment the below line to enable debug prints 
*/
//#define VERBOSE 1

#define MAX_LOGFILE_SIZE (1<<20)
#define MAX_LOGLINE_SIZE 100
#define MAX_PROC_NUM 10
#define MAX_PNAME_LEN 50
#define MAX_NUM_THREADS 100*500
#define MAX_NUM_BLOCKS 100
#define MAX_THREADS_PER_BLOCK 400

/* struct to hold process names read from the proclistfile. */
typedef struct _proc_entry_t {
	char pname[MAX_PNAME_LEN];
	int count;
}proc_entry_t;

/* struct for each thread to put the data calculated by it. */
typedef struct _stats_entry_t {
	proc_entry_t proclist[MAX_PROC_NUM];
}stats_entry_t;

/* CUDA device local func for string copy. */
__device__ void dev_mystrcpy(char *t, char *s)
{
	while ( *s != '\0' ) {
		*t++ = *s++;
	}
	*t = '\0';
}

/* CUDA device local func for getting string length. */
__device__ int dev_my_strlen(char *src)
{
	int len=0;
	while ( *src++ != '\0' )
		len++;
	return (len);
}

/* CUDA device func for comparing strings. */
__device__ int dev_my_strcmp(char *s, char *d)
{
	int len = dev_my_strlen(s), tmplen = dev_my_strlen(d);
	int i=0;

	if (len != tmplen)
		return 1;

	while (i < len) {
		if (*(s+i) != *(d+i))
			return 1;
		i += 1;
	}

	return 0;	
}

/* The global kernel func. 
For the block that a thread is supposed to work with, the below function will calculate the results and populate the corresponding cell
in the dev_stats memory array.
*/
__global__ void dev_calc_stats(char *dev_fileBuf, int *dev_blockStart, int *dev_blockEnd, int numProcs, stats_entry_t *dev_stats, int paddedFileSize, int fileSize)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int i=0, j=0, k=0, bufSize;
	char buf[10000], logline[MAX_LOGLINE_SIZE], tmp[MAX_PNAME_LEN];
	dev_mystrcpy(buf, "");
	dev_mystrcpy(logline, "");
	dev_mystrcpy(tmp, "");

	if (dev_blockStart[idx] > fileSize || dev_blockEnd[idx] > paddedFileSize || dev_blockStart[idx] >= dev_blockEnd[idx])
		return;

	for (i=dev_blockStart[idx] ; i <= paddedFileSize && i <= dev_blockEnd[idx] ; i++) {
		buf[j++] = dev_fileBuf[i];
	}
	buf[j] = '\0';
	bufSize = j;
	
	i = 0; j = 0;
	for (i=0 ; i < bufSize ; i++) {
		if (buf[i] == '\n') {
			if (j <= MAX_LOGLINE_SIZE)
				logline[j] = '\0';
			else {
				j = 0;
				continue;
			}
			k = 0;
			while (k+16 < 100 && logline[k+16] != '[') {
				tmp[k] = logline[k+16];
				k += 1;
			}
			tmp[k] = '\0';
			for (j=0 ; j < numProcs ; j++) {
				if (dev_my_strcmp(dev_stats[idx].proclist[j].pname, tmp) == 0) 
					dev_stats[idx].proclist[j].count += 1;
			}
			j = 0;
		}
		else {
			if (j < MAX_LOGLINE_SIZE)
				logline[j] = buf[i];
			j += 1;
		}
	}

}

__global__ void reducerFunc(stats_entry_t *input_stats,stats_entry_t *output_stats,int numProcesses,int totalThreads)
{
	int j = 0;
	unsigned int myID = blockIdx.x*blockDim.x + threadIdx.x;
	//extern __shared__ stats_entry_t shared_stats[][];

	/*for (j = 0; i < numProcesses; j++)
	{
		shared_stats[tid].proclist[j].count = input_stats[i].proclist[j].count;
	}
	
	__syncthreads();*/

	// do reduction in shared mem
	if (totalThreads > 1 && (myID *2 + 1) < totalThreads)
	{
		for (j = 0; j < numProcesses; j++)
		{
			output_stats[myID].proclist[j].count = input_stats[2*myID].proclist[j].count + input_stats[2*myID + 1].proclist[j].count;
		}
	}
}


int main(int argc, char *argv[])
{
	FILE *fp_logfile, *fp_proclist;
	char *fileBuf=(char *)malloc(sizeof(char)*MAX_LOGFILE_SIZE);
	char *procBuf=(char *)malloc(sizeof(char)*MAX_PROC_NUM*MAX_PNAME_LEN);
	int numThreads=0, numBlocks=0, numThreadsPerBlock=0,  paddedFileSize=0, blockSize=0;
	long fileSize = 0,i = 0;
	int *blockStart=0, *blockEnd=0;
	int numProcs, count, tot_count, pflag=0,done = 0;
	int reducerBlocks, reducerThreadsPerBlock;
	int j, k, start,blockLoop,threadLoop;
	stats_entry_t *stats=0;
	char *pname, proclist[MAX_PROC_NUM][MAX_PNAME_LEN];

	char *dev_fileBuf;
	int *dev_blockStart, *dev_blockEnd;
	stats_entry_t *dev_stats,*dev_reducer_stats;

#ifndef _WIN32
	struct timeval t_start, t_end;
#endif
	hipEvent_t dev_t_start, dev_t_end;
	float time_elapsed;

	if (argc != 3) {
		printf("Usage: ./log_stats path-to-log-file path-to-process-list-file\n");
		exit(1);
	}

	if (!(fp_logfile = fopen(argv[1], "r"))) {
		printf("Error opening Log File!\n");
		exit(1);
	}

	if (!(fp_proclist = fopen(argv[2], "r"))) {
		printf("Error opening Process listing file!\n");
		exit(1);
	}

	/* Read up the proclistfile in a local buffer in memory. */
	i = 0;
	for (i=0 ; !feof(fp_proclist) ; ) {
		i += fread(&(procBuf[i]), 1, 1, fp_proclist);
	}

	/* Read up the entire logfile in a local buffer in memory. */
	i = 0;
	for (i=0 ; !feof(fp_logfile) ; ) {
		i += fread(&(fileBuf[i]), 1, 1, fp_logfile);
	}
	fileSize = i;

#ifdef VERBOSE
	printf("procListFile:\n%s", procBuf);
#endif

	/* Extract out all the process names from proclistfile and populate the proclist array. */
	i = 0;
	pname = strtok(procBuf, "\n");
	while (pname) {
		strcpy(proclist[i], pname);
		i++;
		pname = strtok(NULL, "\n");
	}
	numProcs = i;

#ifdef VERBOSE
	printf("numProcs:%d\n", numProcs);
	for (i=0 ; i < numProcs ; i++) {
		printf("%s\n", proclist[i]);
	}
#endif

	if (fileSize < 65536) {
		hipMalloc((void **)&dev_fileBuf, sizeof(char)*MAX_LOGFILE_SIZE);
		hipMemset((void *)dev_fileBuf, 0, sizeof(char)*MAX_LOGFILE_SIZE);

		hipMalloc((void **)&dev_blockStart, sizeof(int)*MAX_NUM_THREADS);
		hipMemset((void *)dev_blockStart, 0, sizeof(int)*MAX_NUM_THREADS);

		hipMalloc((void **)&dev_blockEnd, sizeof(int)*MAX_NUM_THREADS);
		hipMemset((void *)dev_blockEnd, 0, sizeof(int)*MAX_NUM_THREADS);

		hipMalloc((void **)&dev_stats, sizeof(stats_entry_t)*MAX_NUM_THREADS);
		hipMemset((void *)dev_stats, 0, sizeof(stats_entry_t)*MAX_NUM_THREADS);

		hipMalloc((void **)&dev_reducer_stats, sizeof(stats_entry_t)*MAX_NUM_THREADS);
		hipMemset((void *)dev_reducer_stats, 0, sizeof(stats_entry_t)*MAX_NUM_THREADS);

		blockStart = (int *)malloc(sizeof(int)*MAX_NUM_THREADS);
		blockEnd = (int *)malloc(sizeof(int)*MAX_NUM_THREADS);

		stats = (stats_entry_t *)malloc(sizeof(stats_entry_t)*(MAX_NUM_THREADS));
	}

	for (blockLoop = 1; pow((float)2,blockLoop) <  MAX_NUM_BLOCKS; blockLoop++)
	{
		numBlocks = pow((float)2,blockLoop);

		/* Vary the number of threads per block by some offset. */
		for (threadLoop = 1; pow((float)2,threadLoop) <  MAX_THREADS_PER_BLOCK; threadLoop++)
		{
			numThreadsPerBlock = pow((float)2,threadLoop);

			//numBlocks = 25 ; numThreadsPerBlock = 324;
			/* The actual number of threads to be used for this run of the program. */
			numThreads = numBlocks * numThreadsPerBlock;

			if (fileSize > 65535) {
				blockStart = (int *)malloc(sizeof(int)*MAX_NUM_THREADS);
				blockEnd = (int *)malloc(sizeof(int)*MAX_NUM_THREADS);
				stats = (stats_entry_t *)malloc(sizeof(stats_entry_t)*(MAX_NUM_THREADS));
			}

			for (i=0 ; i < MAX_NUM_THREADS ; i++) {
				blockStart[i] = 0;
			}

			for (i=0 ; i < MAX_NUM_THREADS ; i++) {
				blockEnd[i] = 0;
			}

			for (i=0 ; i < MAX_NUM_THREADS ; i++)  {
				for (j=0 ; j < numProcs ; j++) {
					strcpy(stats[i].proclist[j].pname, "");
					stats[i].proclist[j].count = 0;
				}
			}

			for (i=0 ; i < numThreads ; i++) {
				for (j=0 ; j < numProcs ; j++) {
					strcpy(stats[i].proclist[j].pname, proclist[j]);
					stats[i].proclist[j].count = 0;
				}
			}

			/* Do padding etc. Adjust the length. */
			paddedFileSize = fileSize;
			blockSize = (int)fileSize/numThreads;

			if ( fileSize%numThreads != 0 ) {
				paddedFileSize = fileSize + (numThreads - (fileSize%numThreads));
				blockSize = (int)paddedFileSize/numThreads;
				memset(&(fileBuf[fileSize]), 0, paddedFileSize - fileSize);
			}

			if (blockSize < 20 || blockSize >= 10000) { ;
			/* If the blockSize falls below 20, then no single block can contain any process name. So skip this invocation. 
			Uncomment the below line to display the corresponding message in the program output.
			*/
			/* printf("blockSize:%d numThreads:%d - No legal processing possible for this configuration.!\n", blockSize, numThreads);*/
			continue;
			}
#ifdef VERBOSE
			printf("LogFile:\n%s\n", fileBuf);

			printf("fileSize:%d paddedFileSize:%d blockSize:%d\n\n", fileSize, paddedFileSize, blockSize);
#endif

			int x; 
			//int activeThreads;
			/* Build up blockStart and blockEnd arrays. They will keep track of start and end of every block for this run. */
			for (i=0, k=0, start=0 ; i < numThreads; i++, j++) {

				blockStart[i] = start;
				k = 0;

				if (start+blockSize >= paddedFileSize) {
					blockEnd[i] = paddedFileSize;
					//activeThreads = i;
					for (x = i+1 ; x < numThreads ; x++) {
						blockStart[x] = paddedFileSize;
						blockEnd[x] = paddedFileSize;
					}
					break;
				}

				if (fileBuf[(start+blockSize)] != '\n') {
					k = 1;
					while (((start+blockSize+k) <= paddedFileSize) && (fileBuf[start+blockSize+k] != '\n'))
						k += 1;
					blockEnd[i] = start + blockSize + k;
				} else {
					blockEnd[i] = start + blockSize;
				}

				if (blockEnd[i] > paddedFileSize)
					blockEnd[i] = paddedFileSize;

				if ((blockEnd[i]+1) <= paddedFileSize)
					start = blockEnd[i] + 1;
				else
					start = paddedFileSize;
			}

#ifdef VERBOSE
			printf("Initialized Data as follows:\n");
			for (i=0 ; i < numThreads ; i++) {
				printf("Block %d\n", i);
				printf("blockStart:%d blockEnd:%d\n", blockStart[i], blockEnd[i]);
				for (j=blockStart[i] ; j<blockEnd[i] ; j++) { ;
					printf("%c", fileBuf[j]);
				}
				printf("\nStats:\n");
				for (j=0 ; j < numProcs ; j++) {
					printf("%s %d\n", stats[i].proclist[j].pname, stats[i].proclist[j].count);
				}
				printf("\n\n");
			}
#endif

			if (fileSize > 65536) {
				hipMalloc((void **)&dev_fileBuf, sizeof(char)*MAX_LOGFILE_SIZE);
				hipMemset((void *)dev_fileBuf, 0, sizeof(char)*MAX_LOGFILE_SIZE);

				hipMalloc((void **)&dev_blockStart, sizeof(int)*MAX_NUM_THREADS);
				hipMemset((void *)dev_blockStart, 0, sizeof(int)*MAX_NUM_THREADS);

				hipMalloc((void **)&dev_blockEnd, sizeof(int)*MAX_NUM_THREADS);
				hipMemset((void *)dev_blockEnd, 0, sizeof(int)*MAX_NUM_THREADS);

				hipMalloc((void **)&dev_stats, sizeof(stats_entry_t)*MAX_NUM_THREADS);
				hipMemset((void *)dev_stats, 0, sizeof(stats_entry_t)*MAX_NUM_THREADS);

				hipMalloc((void **)&dev_reducer_stats, sizeof(stats_entry_t)*MAX_NUM_THREADS);
				hipMemset((void *)dev_reducer_stats, 0, sizeof(stats_entry_t)*MAX_NUM_THREADS);

			}

			hipEventCreate(&dev_t_start);
			hipEventCreate(&dev_t_end);
			hipDeviceSynchronize();
#ifndef _WIN32
			gettimeofday(&t_start, NULL);
#endif
			/* Copy the data over to Device's Global Memory. */
			hipMemcpy(dev_fileBuf, fileBuf, sizeof(char)*paddedFileSize, hipMemcpyHostToDevice);
			hipMemcpy(dev_blockStart, blockStart, sizeof(int)*numThreads, hipMemcpyHostToDevice);
			hipMemcpy(dev_blockEnd, blockEnd, sizeof(int)*numThreads, hipMemcpyHostToDevice);
			hipMemcpy(dev_stats, stats, sizeof(stats_entry_t)*numThreads, hipMemcpyHostToDevice);

			hipEventRecord(dev_t_start, 0);

			dev_calc_stats <<< numBlocks, numThreadsPerBlock >>> (dev_fileBuf, dev_blockStart, dev_blockEnd, numProcs, dev_stats, paddedFileSize, fileSize);

			hipEventRecord(dev_t_end, 0);

			hipEventSynchronize(dev_t_end);
			hipEventElapsedTime(&time_elapsed, dev_t_start, dev_t_end );
			hipEventDestroy(dev_t_start);
			hipEventDestroy(dev_t_end);
			hipDeviceSynchronize();

			//hipMemcpy(stats, dev_stats, sizeof(stats_entry_t)*numThreads, hipMemcpyDeviceToHost);

#ifdef VERBOSE
	        	printf("Final Data as follows:\n");
		        for (i=0 ; i < numThreads ; i++) {
        		        printf("Block %d\n", i);
                		printf("blockStart:%d blockEnd:%d\n", blockStart[i], blockEnd[i]);
		                for (j=blockStart[i] ; j<blockEnd[i] ; j++) { ;
        		                printf("%c", fileBuf[j]);
                		}
	                	printf("\nStats:\n");
	        	        for (j=0 ; j < numProcs ; j++) { ;
        	        	        printf("%s %d\n", stats[i].proclist[j].pname, stats[i].proclist[j].count);
                		}
	                	printf("\n\n");
	        	}
#endif
			done = 0;
			reducerBlocks = numBlocks;
			reducerThreadsPerBlock = numThreadsPerBlock;
			hipMemcpy(dev_reducer_stats, dev_stats, sizeof(stats_entry_t)*numThreads, hipMemcpyDeviceToDevice);
			while (done == 0)
			{

				hipMemcpy(stats, dev_stats, sizeof(stats_entry_t)*numThreads, hipMemcpyDeviceToHost);

				/*for (i=0 ; i < numThreads ; i++) {
					printf("\nBefore Reduction: Thread %d - ",i);
					for (j=0 ; j < numProcs ; j++) { 
						printf("%s %d\n", stats[i].proclist[j].pname, stats[i].proclist[j].count);
					}
				}*/

				if (reducerThreadsPerBlock == 1)
				{
					reducerBlocks = reducerBlocks / 2;
				}
				else
				{
					reducerThreadsPerBlock = reducerThreadsPerBlock / 2;
				}
				if (reducerThreadsPerBlock == 1 && reducerBlocks == 1)
				{
					done = 1;
				}
				//printf("Reducing %d,%d\n",reducerBlocks,reducerThreadsPerBlock);

				reducerFunc <<< reducerBlocks,reducerThreadsPerBlock >>> (dev_stats,dev_reducer_stats,numProcs,numBlocks * numThreadsPerBlock);

				hipMemcpy(dev_stats, dev_reducer_stats, sizeof(stats_entry_t)*numThreads, hipMemcpyDeviceToDevice);
			}
			
			hipMemcpy(stats, dev_stats, sizeof(stats_entry_t)*numThreads, hipMemcpyDeviceToHost);
			/*for (i=0 ; i < numThreads ; i++) {
				printf("\nAfter Reduction Thread %d - ",i);
				for (j=0 ; j < numProcs ; j++) { 
					printf("%s %d\n", stats[i].proclist[j].pname, stats[i].proclist[j].count);
				}
			}*/
			/* Aggregate the results calculated by each block. 
			
			tot_count = 0;
			for (j=0 ; j < numProcs ; j++) {
				count = 0;
				for (i=0 ; i < numThreads ; i++) {
					count += stats[i].proclist[j].count;
				}
				if (!pflag)
					printf("pName: %s count: %d\n", stats[0].proclist[j].pname, count);
				tot_count += count;
			}

			if (!pflag)
				printf("Total Number of loglines: %d\n", tot_count);*/

			tot_count = 0;
			for (j=0 ; j < numProcs ; j++) {

				if (!pflag)
					printf("pName: %s count: %d\n", stats[0].proclist[j].pname, stats[0].proclist[j].count);
				tot_count += stats[0].proclist[j].count;
			}

			if (!pflag)
				printf("Total Number of loglines: %d\n", tot_count);

#ifndef _WIN32
			gettimeofday(&t_end, NULL);
			printf("blockSize:%d numThreads:%d totalCount:%d CPUTime:%8ld GPUTime:%f %d %d\n", blockSize, numThreads, tot_count, t_end.tv_usec - t_start.tv_usec + (t_end.tv_sec*1000000 - t_start.tv_sec*1000000),time_elapsed, numBlocks, numThreadsPerBlock);
#else
			printf("blockSize:%d numThreads:%d totalCount:%d GPUTime:%f\n", blockSize, numThreads, tot_count,time_elapsed);
#endif

			if (!pflag)
				pflag = 1;

			if (fileSize > 65536) {
				hipFree(dev_stats);
				hipFree(dev_blockStart);
				hipFree(dev_blockEnd);
				hipFree(dev_fileBuf);
			}

			if (fileSize > 65535) {
				free(blockStart);
				free(blockEnd);
				free(stats);
			}
		}
	}

	return 0;
}

