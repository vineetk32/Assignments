#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "util.h"
#include <pthread.h>

#define MAX_THREADS 20
#define NUM_THREADS 8
#define MAX_PROCESSES 12

struct threadPackage_t
{
	long startOffset;
	long endOffset;
	char **processList;
	int numProcesses;
	int *processCount;
	char *buffer;
};

void *threadFunc(void *threadPackage)
{
	char **splitBuff;
	int i,numTokens;

	struct threadPackage_t *package;
	char *tempBuff;

	long start,end,curr;
	int exitFlag = 0;

	package = (struct threadPackage_t *) threadPackage; 

	start = package->startOffset;
	end = package->endOffset;


	#ifdef __DEBUG
	printf("\nstart: %ld end: %ld curr: %ld - ",start,end,curr);
	#endif

	//Allocate the array which will be used to hold each individual word in a line.
	splitBuff = (char **) malloc ( MAX_WORDS * sizeof(char *));
	tempBuff = (char *) malloc ( BUFFER_SIZE * sizeof(char));

	for(i = 0; i < MAX_WORDS;i++)
	{
		splitBuff[i] = (char *) malloc ( MAX_WORD_LENGTH * sizeof(char));
	}

	//Read the input data between the offsets char-by-char
	while (exitFlag == 0)
	{
		/*while (package->buffer[start] != '\n')
		{
			start++;
			if (package->buffer[start] == '\0')
			{
				exitFlag = 1;
				break;
			}
		}*/
		start++;
		curr = start + 1;
		if (end != -1)
		{
			while (package->buffer[curr] != '\n' && curr < end)
			{
				curr++;
			}

			if (curr >= end)
			{
				exitFlag = 1;
			}
			strncpy(tempBuff,(package->buffer) + start,curr-start);
			curr++;
		}
		else
		{
			while (package->buffer[curr] != '\n' && package->buffer[curr] != '\0')
			{
				curr++;
			}

			if (package->buffer[curr] == '\0')
			{
				exitFlag = 1;
			}
			strncpy(tempBuff,(package->buffer)+ start,curr-start);
		}
		#ifdef __DEBUG
		printf("\nReading %d --> %d (%s)",start,curr,tempBuff);
		#endif
		tempBuff[curr - start - 1] = '\0';
		start = curr;
		if (strlen(tempBuff) == 0)
		{
			#ifdef __DEBUG
			printf("\nZero-len tempBuff! start - %ld curr - %ld end - %ld",start,curr,end);
			#endif
		}
		else
		{
			package->processCount[package->numProcesses]++;
		}

		#ifdef __DEBUG
		//printf("\nTempBuff - %s",tempBuff);
		#endif
	
		//TODO: Apparently, strtok_r is thread-safe only for dynamic mem. Check
		numTokens = splitLine(tempBuff,splitBuff," :[]");
		
		//Check if the current logLine is written by a process we're interested in.
		if (numTokens > 5)
		{
			if ( (i = arrayContains(package->processList,splitBuff[5],package->numProcesses)) > 0)
			{
				package->processCount[i]++;
			}
			//for (i = 0; i < numTokens; i++)
			//{
			//	printf("\t%d: %s",i,splitBuff[i]);
			//}
			//(package->*totalLines)++;
		}
		else
		{
			#ifdef __DEBUG
			printf("\nBad Line - %s",tempBuff);
			#endif
		}
	}
	for (i = 0; i < package->numProcesses; i++)
	{
		free(splitBuff[i]);
	}
	free(splitBuff);
}

int adjustThreadOffsets(char *fileContents,int fileSize,long *offsetArray,int numThreads)
{
	int i,blockSize;
	long ptr = 0;

	blockSize = fileSize / numThreads;
	offsetArray[0] = 0;
	
	//Adjust the offsets so that each thread gets atleast blockSize + EOL to process.
	for (i = 1; i < numThreads; i++)
	{
		//If previous offset is set to EOF, all further offsets must also be set to EOF.
		if (offsetArray[i - 1] != -1)
		{
			//fseek(flog,blockSize,offsetArray[i - 1]);
			ptr += blockSize;
		}
		else
		{
			offsetArray[i] = -1;
		}
		while ( fileContents[ptr] != '\n' && fileContents[ptr] != '\0')
		{
			ptr++;
		}
		
		if ( fileContents[ptr] == '\n')
		{
			offsetArray[i] = ptr;
		}
		else if (fileContents[ptr] == '\0')
		{
			offsetArray[i] = -1;
		}
	}

	#ifdef __DEBUG
	printf("\noffsetArray - ");
	for (i = 0; i < numThreads; i++)
	{
		printf(" %d: %ld",i,offsetArray[i]);
	}
	#endif
}

int main(int argc, char **argv)
{
	FILE *flog,*fproc;
	char tempBuff[BUFFER_SIZE] = {'\0'};
	int i = 0,j = 0,k = 0,numProcesses,totalLines = 0,fileSize;
	char *processList[MAX_PROCESSES];
	int processCount[MAX_THREADS][MAX_PROCESSES];
	long offsetArray[MAX_THREADS];
	struct timespec start_time,end_time;
	pthread_t threads[MAX_THREADS];
	char *fileContents;
	struct threadPackage_t threadPackage[MAX_THREADS];
	int printedOnce = 0,numThreads;


	if (argc != 3)
	{
		printf("Incorrect usage!\nUsage - %s path-to-log-file path-to-process-list-file.\n",argv[0]);
		return -1;
	}

	clock_gettime(CLOCK_MONOTONIC,&start_time);

	fproc = fopen(argv[2],"r");
	if (fproc == NULL)
	{
		printf("\nError opening processfile!\n");
		return -1;
	}
	
	//Read every processName from the processes file
	i = 0;
	while (fscanf(fproc,"%s",tempBuff) >= 0)
	{
		processList[i] = strdup(tempBuff);
		i++;
	}
	fclose(fproc);
	numProcesses = i;
	
	flog = fopen(argv[1],"r");
	if (flog == NULL)
	{
		printf("\nError opening logfile!\n");
		return -1;
	}
	//Get the total byte size of the file.
	if (fseek(flog,0,SEEK_END) == 0)
	{
		fileSize = ftell(flog);
	}

	fileContents = (char *) malloc (sizeof(char) * fileSize);
	rewind(flog);
	fread(fileContents,1,fileSize,flog);
	fclose(flog);

	for (k = 1; (k * 2) <  16; k++)
	{
		numThreads = k * 2;
		for (i = 0; i <= numThreads; i++)
		{
			for (j = 0; j < MAX_PROCESSES; j++)
			{
				processCount[i][j] = 0;
			}
		}

		adjustThreadOffsets(fileContents,fileSize,offsetArray,numThreads);

		//Get the linecount
		/*while ( fgets(tempBuff,1024,flog) > 0)
		{
			totalLines++;
		}
		threadLines = totalLines / NUM_THREADS;
		offsetArray[0] = 0;
		for ( i = 1; i < NUM_THREADS; i++)
		{
			offsetArray[i] = offsetArray[i - 1] + threadLines;
		}*/

		for (i = 0 ;i < numThreads; i++)
		{
			threadPackage[i].processCount = processCount[i];
			threadPackage[i].processList = processList;
			threadPackage[i].numProcesses = numProcesses;
			threadPackage[i].buffer = fileContents;
			threadPackage[i].startOffset = offsetArray[i];

			if (i < (numThreads - 1))
			{
				threadPackage[i].endOffset = offsetArray[i+1] - 1;
			}
			else
			{
				threadPackage[i].endOffset = -1;
			}

			#ifdef __DEBUG
			printf("\nThreadPackage[%d] - \n",i);
			printf("startOffset: %ld endOffset: %ld\n",threadPackage[i].startOffset,threadPackage[i].endOffset);
			#endif
			pthread_create(&threads[i],NULL,&threadFunc,&threadPackage[i]);
		}

		for (i = 0 ;i < numThreads; i++)
		{
			if (pthread_join(threads[i],NULL) != 0)
			{
				printf("\nERROR: pthread_join returned non-zero!\n");
			}
		}

		clock_gettime(CLOCK_MONOTONIC,&end_time);
		//blockSize: numOfThreads: totalCount: runningTime: 

		//Now collate all the process counts received from each threads into one.
		for (i = 0 ; i < numThreads; i++)
		{
			for (j = 0; j < MAX_PROCESSES; j++)
			{
				processCount[numThreads][j] += processCount[i][j];
			}
			totalLines += processCount[i][numProcesses];
		}

		if (printedOnce == 0)
		{
			for(i = 0; i < numProcesses;i++)
			{
				printf("\npName:%s count:%d",processList[i],processCount[numThreads][i]);
			}
			printf("\nTotal Number of loglines: %d\n",totalLines);
			printedOnce = 1;
		}
		//printf("\nNow %d\n",numThreads);
		printf("\n%d: %d: %d: %lfs: \n",fileSize/numThreads,numThreads,totalLines,(end_time.tv_sec - start_time.tv_sec) +  (end_time.tv_nsec - start_time.tv_nsec)/1000000000.0);
	}
	//Be a good boy and release all the memory
	for (i = 0; i < numProcesses; i++)
	{
		free(processList[i]);
	}
	return 0;

}
