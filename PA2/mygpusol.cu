#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

//#define __DEBUG

#define element_addr(a, m, n, d) (a + ((m) * (d) + n))
#define element(a, m, n, d) (((m >= 0)&&(m < d)&&(n >= 0)&&(n < d))? (a[(m) * (d) + n]) : 0) 

#define CUDA_CALL(cmd) do { \
	if((err = cmd) != hipSuccess) { \
		printf("(%d) Cuda Error:(%d) %s\n", __LINE__,int(err), hipGetErrorString(err) ); \
	} \
} while(0)


#define BLK_SZ 16
#define BLK_SIDE 4

__global__ void computeKernel(int *living, float *honeys[2], float *honeyr, int d, float rbee, float rflow) {
	//honeyr[threadIdx.x] = honeys[0][threadIdx.x];
	//honeyr[threadIdx.x] = threadIdx.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	//honeyr[i*d + j] = i+j;
	*(element_addr(honeyr, i, j, d)) = element(honeyr,i-1,j-1,d);
}


__global__ void computeKernelReal(int *living, float *honeyin,float *honeyout, int d, float rbee, float rflow) {
	
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	*(element_addr(honeyin, i, j, d)) = rflow * (element(honeyin, i-1, j-1, d) + element(honeyin, i-1, j, d) + element(honeyin, i-1, j+1, d) + element(honeyin, i, j-1, d)   + element(honeyin, i, j+1, d) + element(honeyin, i+1, j-1, d) + element(honeyin, i+1, j, d) + element(honeyin, i+1, j+1, d) ) + (1.0 - 8.0 * rflow) * element(honeyin, i, j, d) + rbee * element(living, i, j, d);
} 

int calculateGPU(const int *living, float *honey[2], int d, int n, float rbee, float rflow)
{
	hipError_t err;
	clock_t start, end;
	hipEvent_t kstart, kstop;
	float ktime;
	double time;

	int i;


	/* PA2: Define your local variables here */
	int *living_d;
	float *honeyin_d;
	float *honey_r;

	/* Set up device timers */
	#line 49
	CUDA_CALL(hipSetDevice(0));
	#line 50
	CUDA_CALL(hipEventCreate(&kstart));
	#line 53
	CUDA_CALL(hipEventCreate(&kstop));

	/* Start GPU end-to-end timer */
	start = clock();

	/* PA2: Add CUDA kernel call preparation code here */
	#line 60
	CUDA_CALL(hipMalloc((void **)&living_d, d * d * sizeof(int)));
	#line 62
	CUDA_CALL(hipMalloc((void **)&honeyin_d, d * d * sizeof(float)));
	#line 66
	CUDA_CALL(hipMalloc((void **)&honey_r, d * d * sizeof(float)));
	#line 68
	CUDA_CALL(hipMemcpy(living_d, living, d * d * sizeof(int), hipMemcpyHostToDevice));
	#line 70
	CUDA_CALL(hipMemcpy(honeyin_d, honey[0], d * d * sizeof(float), hipMemcpyHostToDevice));

	/* Start GPU computation timer */
	#line 76
	CUDA_CALL(hipEventRecord(kstart, 0));

	/* PA2: Add main honey level simulation loop here */
	dim3 dimGrid(d/BLK_SIDE,d/BLK_SIDE);
	dim3 dimBlock(BLK_SIDE,BLK_SIDE);
	for (i=0;i< n;i++)
	{
		//printf("Calling kernel with resin-%d and src-%d\n",resin,src);
		computeKernelReal<<<dimGrid,dimBlock>>>(living_d,honeyin_d,honey_r,d,rbee,rflow);
		#line 4848
		//CUDA_CALL(cudaThreadSynchronize());
		CUDA_CALL(hipMemcpy(honeyin_d,honey_r,d * d * sizeof(float),hipMemcpyDeviceToDevice ));
	}

	//computeKernel<<<dimGrid,dimBlock>>>(living_d,honey_d,honey_r,d,rbee,rflow);


	/* Stop GPU computation timer */
	#line 88
	CUDA_CALL(hipEventRecord(kstop, 0));
	#line 90
	CUDA_CALL(hipEventSynchronize(kstop));

	#line 93
	CUDA_CALL(hipEventElapsedTime(&ktime, kstart, kstop));
	printf("GPU computation: %f msec\n", ktime);



	/* PA2: Add post CUDA kernel call processing and cleanup here */
	#line 98
	//CUDA_CALL(cudaMemcpy(honey[0],honey_d[resin],d * d * sizeof(float),cudaMemcpyDeviceToHost));
	CUDA_CALL(hipMemcpy(honey[1],honey_r,d * d * sizeof(float),hipMemcpyDeviceToHost));

	printf("\nhoney[] after cuda kernel call -\n");
	for(int i = 0; i < d; i++ ) {
		for(int j = 0; j < d; j++ ) {
			printf("%f ", element(honey[1], i, j, d));
		}
		printf("\n");
	}
	
	#line 100
	CUDA_CALL(hipFree(living_d));
	#line 102
	CUDA_CALL(hipFree(honeyin_d));
	#line 106
	CUDA_CALL(hipFree(honey_r));

	/* Stop GPU end-to-end timer and timer cleanup */
	end = clock();
	#line 111
	CUDA_CALL(hipEventDestroy(kstart));
	#line 113
	CUDA_CALL(hipEventDestroy(kstop));
	time = ((double)(end-start))/CLOCKS_PER_SEC;
	printf("GPU end-to-end: %lf sec\n", time);
	return 1;
}
